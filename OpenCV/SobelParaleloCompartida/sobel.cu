#include "hip/hip_runtime.h"
#include <cv.h>
#include <cv.h>
#include <highgui.h>
#include <time.h>
#include <hip/hip_runtime.h>
#include <math.h>
#include <time.h>

#define RED 2
#define GREEN 1
#define BLUE 0

#define MASK_WIDTH 3
#define TILE_SIZE 32

__constant__ char d_mask[MASK_WIDTH*MASK_WIDTH];
__constant__ char d_maskt[MASK_WIDTH*MASK_WIDTH];

using namespace cv;

__device__ unsigned char clamp(int value){
    if(value < 0)
        value = 0;
    else
        if(value > 255)
            value = 255;
    return (unsigned char)value;
}

__global__ void gray(unsigned char *imageNormal, int width, int height, unsigned char *imageGray){
	int row = blockIdx.y*blockDim.y+threadIdx.y;
	int col = blockIdx.x*blockDim.x+threadIdx.x;

	if((row < height) && (col < width)){
		imageGray[row*width+col] = imageNormal[(row*width+col)*3+RED]*0.299 + imageNormal[(row*width+col)*3+GREEN]*0.587 + imageNormal[(row*width+col)*3+BLUE]*0.114;
	}
}

__global__ void sobelX(unsigned char *imageGray, int width, int height, unsigned int maskWidth, unsigned char *imageSobel){
	__shared__ float N_ds[TILE_SIZE + MASK_WIDTH - 1][TILE_SIZE+ MASK_WIDTH - 1];			//se establecen la submatriz y queda en memoria compartida
																																										//el tamaño del array en memoria global debe ser mas largo que el vector normal para darle espacio a los elementos de la izquierda, centro y derecha en total es TILE_SIZE + MASK_WIDTH - 1
    int n = maskWidth/2;
    
    //------Cargar los elementos de la matriz de la matriz de entrada en memoria compartida------
    //Cargar elementos izquierda derecha
    int dest = threadIdx.y*TILE_SIZE+threadIdx.x;
	int destY = dest / (TILE_SIZE+MASK_WIDTH-1);
	int destX = dest % (TILE_SIZE+MASK_WIDTH-1);
    int srcY = blockIdx.y * TILE_SIZE + destY - n;
	int srcX = blockIdx.x * TILE_SIZE + destX - n;
    int src = (srcY * width + srcX);
	
    if (srcY >= 0 && srcY < height && srcX >= 0 && srcX < width)		//si srcY es negativo son elementos fantasmas, si srcX es negativo son elementos fantasmas
        N_ds[destY][destX] = imageGray[src];
    else
        N_ds[destY][destX] = 0;					//asigna en 0 los elementos fantasmas

    //Cargar elementos del centro
    dest = threadIdx.y * TILE_SIZE + threadIdx.x + TILE_SIZE * TILE_SIZE;
    destY = dest /(TILE_SIZE + MASK_WIDTH - 1);
	destX = dest % (TILE_SIZE + MASK_WIDTH - 1);
    srcY = blockIdx.y * TILE_SIZE + destY - n;
    srcX = blockIdx.x * TILE_SIZE + destX - n;
    src = (srcY * width + srcX);
	
    if (destY < TILE_SIZE + MASK_WIDTH - 1) {
        if (srcY >= 0 && srcY < height && srcX >= 0 && srcX < width)
            N_ds[destY][destX] = imageGray[src];
        else
            N_ds[destY][destX] = 0;
    }
    __syncthreads();
    //------Termina de cargar los elementos de la matriz de la matriz de entrada en memoria compartida------

		//-----llenamos la matriz de salida
    int accum = 0;
    int y, x;
    for (y = 0; y < maskWidth; y++)
        for (x = 0; x < maskWidth; x++)
            accum += N_ds[threadIdx.y + y][threadIdx.x + x] * d_mask[y * maskWidth + x];
    y = blockIdx.y * TILE_SIZE + threadIdx.y;
    x = blockIdx.x * TILE_SIZE + threadIdx.x;
    if (y < height && x < width)
        imageSobel[(y * width + x)] = clamp(accum);
    __syncthreads();
    //-----terminamos de llenar la matriz de salida

}

__global__ void sobelY(unsigned char *imageGray, int width, int height, unsigned int maskWidth, unsigned char *imageSobel){
	__shared__ float N_ds[TILE_SIZE + MASK_WIDTH - 1][TILE_SIZE+ MASK_WIDTH - 1];			//se establecen la submatriz y queda en memoria compartida
																																										//el tamaño del array en memoria global debe ser mas largo que el vector normal para darle espacio a los elementos de la izquierda, centro y derecha en total es TILE_SIZE + MASK_WIDTH - 1
    int n = maskWidth/2;
    
    //------Cargar los elementos de la matriz de la matriz de entrada en memoria compartida------
    //Cargar elementos izquierda derecha
    int dest = threadIdx.y*TILE_SIZE+threadIdx.x;
	int destY = dest / (TILE_SIZE+MASK_WIDTH-1);
	int destX = dest % (TILE_SIZE+MASK_WIDTH-1);
    int srcY = blockIdx.y * TILE_SIZE + destY - n;
	int srcX = blockIdx.x * TILE_SIZE + destX - n;
    int src = (srcY * width + srcX);
	
    if (srcY >= 0 && srcY < height && srcX >= 0 && srcX < width)		//si srcY es negativo son elementos fantasmas, si srcX es negativo son elementos fantasmas
        N_ds[destY][destX] = imageGray[src];
    else
        N_ds[destY][destX] = 0;					//asigna en 0 los elementos fantasmas

    //Cargar elementos del centro
    dest = threadIdx.y * TILE_SIZE + threadIdx.x + TILE_SIZE * TILE_SIZE;
    destY = dest /(TILE_SIZE + MASK_WIDTH - 1);
	destX = dest % (TILE_SIZE + MASK_WIDTH - 1);
    srcY = blockIdx.y * TILE_SIZE + destY - n;
    srcX = blockIdx.x * TILE_SIZE + destX - n;
    src = (srcY * width + srcX);
	
    if (destY < TILE_SIZE + MASK_WIDTH - 1) {
        if (srcY >= 0 && srcY < height && srcX >= 0 && srcX < width)
            N_ds[destY][destX] = imageGray[src];
        else
            N_ds[destY][destX] = 0;
    }
    __syncthreads();
    //------Termina de cargar los elementos de la matriz de la matriz de entrada en memoria compartida------

		//-----llenamos la matriz de salida
    int accum = 0;
    int y, x;
    for (y = 0; y < maskWidth; y++)
        for (x = 0; x < maskWidth; x++)
            accum += N_ds[threadIdx.y + y][threadIdx.x + x] * d_maskt[y * maskWidth + x];
    y = blockIdx.y * TILE_SIZE + threadIdx.y;
    x = blockIdx.x * TILE_SIZE + threadIdx.x;
    if (y < height && x < width)
        imageSobel[(y * width + x)] = clamp(accum);
    __syncthreads();
    //-----terminamos de llenar la matriz de salida

}

__global__ void sobel(unsigned char *imageSobelX, unsigned char *imageSobelY, int width, int height, unsigned char *imageSobel){
	unsigned int row = blockIdx.y*blockDim.y+threadIdx.y;
	unsigned int col = blockIdx.x*blockDim.x+threadIdx.x;

	if((row < height) && (col < width)){
		imageSobel[row* width+col] = __powf((__powf(imageSobelX[row*width+col],2) + __powf(imageSobelY[row*width+col],2)),0.5);
	}

}


int main(int argc, char **argv){
	char h_mask[] = {-1,0,1,-2,0,2,-1,0,1};
	char h_maskt[] = {-1,-2,-1,0,0,0,1,2,1};
	char *imageName = argv[1];
	unsigned char *h_imageNormal, *d_imageNormal, *d_imageGray, *h_imageGray; 
	unsigned char *h_imageSobel, *d_imageSobel, *d_imageSobelX, *d_imageSobelY;

	Mat image;
	image = imread(imageName,1);

	Size s = image.size();
	int width = s.width;
	int height = s.height;
	int size = sizeof(unsigned char)*width*height*image.channels();
	int sizeGray = sizeof(unsigned char)*width*height;

	h_imageNormal = (unsigned char*)malloc(size);
	h_imageGray = (unsigned char*)malloc(sizeGray);
	h_imageSobel = (unsigned char*)malloc(sizeGray);

	hipMalloc((void**)&d_imageNormal,size);
	hipMalloc((void**)&d_imageGray,sizeGray);
	hipMalloc((void**)&d_imageSobel,sizeGray);
	hipMalloc((void**)&d_imageSobelX,sizeGray);
	hipMalloc((void**)&d_imageSobelY,sizeGray);

	h_imageNormal = image.data;
	
	clock_t start = clock(); 

	hipMemcpy(d_imageNormal,h_imageNormal,size, hipMemcpyHostToDevice);
	
	hipMemcpyToSymbol(HIP_SYMBOL(d_mask),h_mask,sizeof(char)*MASK_WIDTH*MASK_WIDTH);
	hipMemcpyToSymbol(HIP_SYMBOL(d_maskt),h_maskt,sizeof(char)*MASK_WIDTH*MASK_WIDTH);
	

	int blockSize = 32;
	dim3 dimBlock(blockSize,blockSize,1);
	dim3 dimGrid(ceil(width/float(blockSize)),ceil(height/float(blockSize)),1);

	gray<<<dimGrid,dimBlock>>>(d_imageNormal,width,height,d_imageGray);
	hipDeviceSynchronize();
	sobelX<<<dimGrid,dimBlock>>>(d_imageGray,width,height,MASK_WIDTH,d_imageSobelX);
	sobelY<<<dimGrid,dimBlock>>>(d_imageGray,width,height,MASK_WIDTH,d_imageSobelY);
	sobel<<<dimGrid,dimBlock>>>(d_imageSobelX,d_imageSobelY,width,height,d_imageSobel);
	
	hipMemcpy(h_imageGray,d_imageGray,sizeGray,hipMemcpyDeviceToHost);
	hipMemcpy(h_imageSobel,d_imageSobel,sizeGray,hipMemcpyDeviceToHost);

	printf("%f;\n", ((double)clock() - start) / CLOCKS_PER_SEC);

	  Mat gray_image;
    gray_image.create(height,width,CV_8UC1);
    gray_image.data = h_imageGray;

    Mat sobel_image;
    sobel_image.create(height,width,CV_8UC1);
    sobel_image.data = h_imageSobel;

    namedWindow(imageName, CV_WINDOW_AUTOSIZE);
    namedWindow("Gray Image CUDA", CV_WINDOW_AUTOSIZE);
    namedWindow("Sobel Image OpenCV", CV_WINDOW_AUTOSIZE);

    imshow(imageName,image);
    imshow("Gray Image CUDA", gray_image);
    imshow("Sobel Image OpenCV",sobel_image);

    waitKey(0);

    hipFree(d_imageNormal);
    hipFree(d_imageGray);
    hipFree(d_imageSobel);
    hipFree(d_imageSobelX);
    hipFree(d_imageSobelY);
    hipFree(d_mask);
    hipFree(d_maskt);

    //free(h_imageNormal);
    //free(h_imageGray);
    //free(h_imageSobel);

    return 0;
}
