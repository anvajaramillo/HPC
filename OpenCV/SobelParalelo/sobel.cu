#include "hip/hip_runtime.h"
#include <cv.h>
#include <cv.h>
#include <highgui.h>
#include <time.h>
#include <hip/hip_runtime.h>
#include <math.h>

#define RED 2
#define GREEN 1
#define BLUE 0

using namespace cv;

__device__ unsigned char clamp(int value){
    if(value < 0)
        value = 0;
    else
        if(value > 255)
            value = 255;
    return (unsigned char)value;
}

__global__ void gray(unsigned char *imageNormal, int width, int height, unsigned char *imageGray){
	int row = blockIdx.y*blockDim.y+threadIdx.y;
	int col = blockIdx.x*blockDim.x+threadIdx.x;

	if((row < height) && (col < width)){
		imageGray[row*width+col] = imageNormal[(row*width+col)*3+RED]*0.299 + imageNormal[(row*width+col)*3+GREEN]*0.587 + imageNormal[(row*width+col)*3+BLUE]*0.114;
	}
}

__global__ void sobelX_Y(unsigned char *imageGray, int width, int height, unsigned int maskWidth, char *mask, unsigned char *imageSobel){
	unsigned int row = blockIdx.y*blockDim.y+threadIdx.y;
	unsigned int col = blockIdx.x*blockDim.x+threadIdx.x;

	int value = 0;
	int n_start_point_row = row - (maskWidth/2);
	int n_start_point_col = col - (maskWidth/2);

	for(int i = 0; i < maskWidth; i++){
		for(int j = 0; j < maskWidth; j++){
			if((n_start_point_col+j >= 0 && n_start_point_col+j < width) && (n_start_point_row+i >= 0 && n_start_point_row+i < height)){
				value += imageGray[(n_start_point_row+i)*width+(n_start_point_col+j)] * mask[i*maskWidth+j];
			}
		}
	}
	imageSobel[row*width+col] = clamp(value);

}

__global__ void sobel(unsigned char *imageSobelX, unsigned char *imageSobelY, int width, int height, unsigned char *imageSobel){
	unsigned int row = blockIdx.y*blockDim.y+threadIdx.y;
	unsigned int col = blockIdx.x*blockDim.x+threadIdx.x;

	if((row < height) && (col < width)){
		imageSobel[row* width+col] = __powf((__powf(imageSobelX[row*width+col],2) + __powf(imageSobelY[row*width+col],2)),0.5);
	}

}


int main(int argc, char **argv){
	char h_mask[] = {-1,0,1,-2,0,2,-1,0,1};
	char h_maskt[] = {-1,-2,-1,0,0,0,1,2,1};
	char *d_mask, *d_maskt;
	char *imageName = argv[1];
	unsigned char *h_imageNormal, *d_imageNormal, *d_imageGray, *h_imageGray; 
	unsigned char *h_imageSobel, *d_imageSobel, *d_imageSobelX, *d_imageSobelY;

	Mat image;
	image = imread(imageName,1);

	Size s = image.size();
	int width = s.width;
	int height = s.height;
	int size = sizeof(unsigned char)*width*height*image.channels();
	int sizeGray = sizeof(unsigned char)*width*height;

	h_imageNormal = (unsigned char*)malloc(size);
	h_imageGray = (unsigned char*)malloc(sizeGray);
	h_imageSobel = (unsigned char*)malloc(sizeGray);

	hipMalloc((void**)&d_imageNormal,size);
	hipMalloc((void**)&d_imageGray,sizeGray);
	hipMalloc((void**)&d_imageSobel,sizeGray);
	hipMalloc((void**)&d_imageSobelX,sizeGray);
	hipMalloc((void**)&d_imageSobelY,sizeGray);
	hipMalloc((void**)&d_mask,sizeof(char)*9);
	hipMalloc((void**)&d_maskt,sizeof(char)*9);

	h_imageNormal = image.data;

	hipMemcpy(d_imageNormal,h_imageNormal,size, hipMemcpyHostToDevice);
	hipMemcpy(d_mask,h_mask,sizeof(char)*9,hipMemcpyHostToDevice);
	hipMemcpy(d_maskt,h_maskt,sizeof(char)*9,hipMemcpyHostToDevice);

	int blockSize = 32;
	dim3 dimBlock(blockSize,blockSize,1);
	dim3 dimGrid(ceil(width/float(blockSize)),ceil(height/float(blockSize)),1);

	gray<<<dimGrid,dimBlock>>>(d_imageNormal,width,height,d_imageGray);
	hipDeviceSynchronize();
	sobelX_Y<<<dimGrid,dimBlock>>>(d_imageGray,width,height,3,d_mask,d_imageSobelX);
	sobelX_Y<<<dimGrid,dimBlock>>>(d_imageGray,width,height,3,d_maskt,d_imageSobelY);
	sobel<<<dimGrid,dimBlock>>>(d_imageSobelX,d_imageSobelY,width,height,d_imageSobel);
	
	hipMemcpy(h_imageGray,d_imageGray,sizeGray,hipMemcpyDeviceToHost);
	hipMemcpy(h_imageSobel,d_imageSobel,sizeGray,hipMemcpyDeviceToHost);

	Mat gray_image;
    gray_image.create(height,width,CV_8UC1);
    gray_image.data = h_imageGray;

    Mat sobel_image;
    sobel_image.create(height,width,CV_8UC1);
    sobel_image.data = h_imageSobel;

    namedWindow(imageName, CV_WINDOW_AUTOSIZE);
    namedWindow("Gray Image CUDA", CV_WINDOW_AUTOSIZE);
    namedWindow("Sobel Image OpenCV", CV_WINDOW_AUTOSIZE);

    imshow(imageName,image);
    imshow("Gray Image CUDA", gray_image);
    imshow("Sobel Image OpenCV",sobel_image);

    waitKey(0);

    hipFree(d_imageNormal);
    hipFree(d_imageGray);
    hipFree(d_imageSobel);
    hipFree(d_imageSobelX);
    hipFree(d_imageSobelY);
    hipFree(d_mask);
    hipFree(d_maskt);

    free(h_imageNormal);
    free(h_imageGray);
    free(h_imageSobel);

    return 0;
}
