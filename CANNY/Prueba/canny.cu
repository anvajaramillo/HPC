#include "hip/hip_runtime.h"
#include <cv.h>
#include <highgui.h>
#include <math.h>
#include <stdio.h>
#include <unistd.h> 
#include <stdlib.h> 
#include <iostream>
#include <string>
#include <hip/hip_runtime.h>

using namespace cv;

#define RED 2
#define GREEN 1
#define BLUE 0

#define MASK_WIDTH_M 3
#define MASK_WIDTH_S 5

#define TILE_SIZE 32	//tamaño de las submatrices
    
__constant__ char d_M[MASK_WIDTH_M*MASK_WIDTH_M];
__constant__ char d_Mt[MASK_WIDTH_M*MASK_WIDTH_M];
__constant__ char d_S[MASK_WIDTH_S*MASK_WIDTH_S];

__device__ unsigned char clamp(int value)
{
    if(value < 0)
        value = 0;
    else
        if(value > 255)
            value = 255;
    return (unsigned char)value;
}

__global__ void img2gray(unsigned char *imgOutput, unsigned char *imgInput, int width, int height)
{
    int row = blockIdx.y*blockDim.y+threadIdx.y;
    int col = blockIdx.x*blockDim.x+threadIdx.x;

    if((row < height) && (col < width)){
        imgOutput[row * width + col] = imgInput[(row * width + col) * 3 + RED] * 0.299 + imgInput[(row * width + col) * 3 + GREEN] * 0.587 + imgInput[(row * width + col) * 3 + BLUE] * 0.114;
    }   
    
}

__global__ void gauss(unsigned char *imgOutput, int maskWidth, unsigned char *imgInput, int width, int height)
{
    int row = blockIdx.y*blockDim.y+threadIdx.y;
    int col = blockIdx.x*blockDim.x+threadIdx.x;
    
	int Pvalue = 0;
    int N_start_point_row = row - (maskWidth/2);
    int N_start_point_col = col - (maskWidth/2);

    if((row < height) && (col < width)){
        for(int i = 0; i < maskWidth; i++){
		  		for(int j = 0; j < maskWidth; j++ ){
		  		    if((N_start_point_col + j >=0 && N_start_point_col + j < width)
		  		            &&(N_start_point_row + i >=0 && N_start_point_row + i < height)){
		  		        Pvalue += imgInput[(N_start_point_row + i)*width+(N_start_point_col + j)] * d_S[i*maskWidth+j];
		  		    }
		  		}
        }
        imgOutput[row * width + col] = clamp(Pvalue/159);
    }
}

__global__ void sobelGradX(unsigned char *imgOutput, int maskWidth, unsigned char *imgInput, int width, int height)
{
	__shared__ float N_ds[TILE_SIZE][TILE_SIZE]; //se establecen la submatriz y queda en memoria compartida		
		
    int y = blockIdx.y * TILE_SIZE + threadIdx.y;
    int x = blockIdx.x * TILE_SIZE + threadIdx.x;
    
    int bx = blockIdx.x;
    int by = blockIdx.y;
    int tx = threadIdx.x;
    int ty = threadIdx.y;

    int row = by * TILE_SIZE + ty;
    int col = bx * TILE_SIZE + tx;
    
    int Pvalue = 0;
    int N_start_point_row = row - (maskWidth/2);
    int N_start_point_col = col - (maskWidth/2);

    for(int m = 0; m < col / TILE_SIZE; m=m+1){
    	
    	N_ds[ty][tx] = imgInput[row*width + m*TILE_SIZE + tx];
    	__syncthreads(); 
    
        for(int i = 0; i < maskWidth; i++){
					for(int j = 0; j < maskWidth; j++ ){
							if((N_start_point_col + j >=0 && N_start_point_col + j < width)
							        &&(N_start_point_row + i >=0 && N_start_point_row + i < height)){						    
							    Pvalue += N_ds[N_start_point_row + i][N_start_point_col + j] * d_M[i*maskWidth+j];
							}
					}
				}	
				if (y < height && x < width)
					imgOutput[(y * width + x)] = clamp(Pvalue);
				__syncthreads(); 
    }    
    
}

__global__ void sobelGradY(unsigned char *imgOutput, int maskWidth, unsigned char *imgInput, int width, int height)
{
    int row = blockIdx.y*blockDim.y+threadIdx.y;
    int col = blockIdx.x*blockDim.x+threadIdx.x;

    int Pvalue = 0;
    int N_start_point_row = row - (maskWidth/2);
    int N_start_point_col = col - (maskWidth/2);

    if((row < height) && (col < width)){
        for(int i = 0; i < maskWidth; i++){
					for(int j = 0; j < maskWidth; j++ ){
							if((N_start_point_col + j >=0 && N_start_point_col + j < width)
							        &&(N_start_point_row + i >=0 && N_start_point_row + i < height)){
							    Pvalue += imgInput[(N_start_point_row + i)*width+(N_start_point_col + j)] * d_Mt[i*maskWidth+j];
							}
					}
		}	
		imgOutput[row * width + col] = clamp(Pvalue);
    }    
    
}

__global__ void sobelFilter(unsigned char *imgSobel, unsigned char *sobelOutputX, unsigned char *sobelOutputY, int width, int height)
{
    int row = blockIdx.y*blockDim.y+threadIdx.y;
    int col = blockIdx.x*blockDim.x+threadIdx.x;

    if((row < height) && (col < width)){
        imgSobel[row * width + col] = __powf((__powf(sobelOutputX[row * width + col],2) + __powf(sobelOutputY[row * width + col],2)), 0.5) ;

    }  
    
}

__global__ void NoSupreMax(int width, int height, unsigned char *imgSobel,unsigned char *nosupmax){
	int row = blockIdx.y*blockDim.y+threadIdx.y;
    int col = blockIdx.x*blockDim.x+threadIdx.x;

    if((row < height) && (col < width)){
		if((imgSobel[row * width + col-1] < imgSobel[row * width + col]) && (imgSobel[row * width + col]< imgSobel[row * width + col+1])){
			nosupmax[row * width + col] = imgSobel[row * width + col];
		}else{
			nosupmax[row * width + col] = 0;
		}
	}	
	

}

int main(int argc, char **argv)
{
    clock_t start, end;
    double cpu_time_used;

    char *imageName = argv[1];
    char h_M[] = {-1,0,1,-2,0,2,-1,0,1};
    char h_Mt[] = {-1,-2,-1,0,0,0,1,2,1};
    char h_S[] = {2,4,5,4,2,4,9,12,9,4,5,12,15,12,5,4,9,12,9,4,2,4,5,4,2};
    unsigned char *h_dataRawImage, *h_imgOutput, *h_imgSobel, *h_suavizada, *h_nosupmax;
    unsigned char *d_dataRawImage, *d_imgOutput, *d_imgSobel, *d_suavizada, *d_nosupmax, *d_sobelOutputX, *d_sobelOutputY;
    
    Mat image;
    image = imread(imageName, 1);
  
    Size img_size = image.size();

    int width = img_size.width;
    int height = img_size.height;
    int size = sizeof(unsigned char) * width * height * image.channels();
    int sizeGray = sizeof(unsigned char) * width * height;

    h_dataRawImage = (unsigned char*)malloc(size);
    h_imgOutput = (unsigned char*)malloc(sizeGray);
    h_suavizada = (unsigned char*)malloc(sizeGray);
    h_imgSobel = (unsigned char*)malloc(sizeGray);
    h_nosupmax = (unsigned char*)malloc(sizeGray);

    hipMalloc((void**)&d_dataRawImage,size);
    hipMalloc((void**)&d_imgOutput,sizeGray);
    hipMalloc((void**)&d_suavizada,sizeGray);
    hipMalloc((void**)&d_imgSobel,sizeGray);
    hipMalloc((void**)&d_nosupmax,sizeGray);
    hipMalloc((void**)&d_sobelOutputX,sizeGray);
    hipMalloc((void**)&d_sobelOutputY,sizeGray);
    hipMalloc((void**)&d_M,sizeof(char)*9);
    hipMalloc((void**)&d_Mt,sizeof(char)*9);
    hipMalloc((void**)&d_S,sizeof(char)*25);

    h_dataRawImage = image.data;
	
    start = clock();

    hipMemcpy(d_dataRawImage ,h_dataRawImage ,size, hipMemcpyHostToDevice);
    
    hipMemcpyToSymbol(HIP_SYMBOL(d_M),h_M,sizeof(char)*MASK_WIDTH_M*MASK_WIDTH_M);
    hipMemcpyToSymbol(HIP_SYMBOL(d_Mt),h_Mt,sizeof(char)*MASK_WIDTH_M*MASK_WIDTH_M);
    hipMemcpyToSymbol(HIP_SYMBOL(d_S),h_S,sizeof(char)*MASK_WIDTH_S*MASK_WIDTH_S);

    int blockSize = 32;
    dim3 dimBlock(blockSize,blockSize,1);
    dim3 dimGrid(ceil(width/float(blockSize)),ceil(height/float(blockSize)),1);
    
    //Escala de Grises
    img2gray<<<dimGrid,dimBlock>>>(d_imgOutput, d_dataRawImage, width, height);
    hipDeviceSynchronize();

    //Suavizado
    gauss<<<dimGrid,dimBlock>>>(d_suavizada, MASK_WIDTH_S, d_imgOutput, width, height);

    // Gradient X
	sobelGradX<<<dimGrid,dimBlock>>>(d_sobelOutputX, MASK_WIDTH_M, d_suavizada, width, height);
		
	// Gradient Y
	sobelGradY<<<dimGrid,dimBlock>>>(d_sobelOutputY, MASK_WIDTH_M, d_suavizada, width, height);

	// Gradient Magnitude
    sobelFilter<<<dimGrid,dimBlock>>>(d_imgSobel, d_sobelOutputX, d_sobelOutputY, width, height);

	//Supresión Máxima
	NoSupreMax<<<dimGrid,dimBlock>>>(width,height,d_imgSobel,d_nosupmax);

    hipMemcpy(h_imgOutput,d_imgOutput,sizeGray,hipMemcpyDeviceToHost);
    hipMemcpy(h_suavizada,d_suavizada,sizeGray,hipMemcpyDeviceToHost);
    hipMemcpy(h_imgSobel,d_imgSobel,sizeGray,hipMemcpyDeviceToHost);
    hipMemcpy(h_nosupmax,d_nosupmax,sizeGray,hipMemcpyDeviceToHost);

    end = clock();

    cpu_time_used = ((double) (end - start)) / CLOCKS_PER_SEC;

    printf("%.10f\n",cpu_time_used);
	
    Mat gray_image;
    gray_image.create(height,width,CV_8UC1);
    gray_image.data = h_imgOutput;
	
    Mat suav_image;
    suav_image.create(height,width,CV_8UC1);
    suav_image.data = h_suavizada;
    
    Mat sobel_image;
    sobel_image.create(height,width,CV_8UC1);
    sobel_image.data = h_imgSobel;
    
    Mat nosupmax_image;
    nosupmax_image.create(height,width,CV_8UC1);
    nosupmax_image.data = h_nosupmax;
    
    namedWindow(imageName, WINDOW_NORMAL);
    namedWindow("Gray Image", WINDOW_NORMAL);
    namedWindow("Gray Image Suavizada", WINDOW_NORMAL);
    namedWindow("Sobel Image", WINDOW_NORMAL);
    namedWindow("No Supesion Image", WINDOW_NORMAL);
    
    imshow(imageName,image);
    imshow("Gray Image", gray_image);
    imshow("Gray Image Suavizada", suav_image);
    imshow("Sobel Image", sobel_image);
    imshow("No Supesion Image", nosupmax_image);

    waitKey(0); 

    hipFree(d_dataRawImage); 
    hipFree(d_imgOutput); 
    hipFree(d_imgSobel); 
    hipFree(d_suavizada);
    hipFree(d_nosupmax);
    hipFree(d_sobelOutputX);
    hipFree(d_sobelOutputY);
    hipFree(d_M);
    hipFree(d_Mt);
    hipFree(d_S);

    return 0;
}
